#include "hip/hip_runtime.h"
#include "optix.h"
#include "RendererObjects.hpp"

rtDeclareVariable( PerRayData_radiance, prd_radiance,      rtPayload, );

rtDeclareVariable( optix::Ray,          ray,               rtCurrentRay, );
rtDeclareVariable( uint2,               launch_index,      rtLaunchIndex, );

rtDeclareVariable( unsigned int,        radiance_ray_type, , );
rtDeclareVariable( float,               scene_epsilon,     , );
rtDeclareVariable( rtObject,            top_object,        , );


//
// Pinhole camera implementation
//
rtDeclareVariable( float3, eye,       , );
rtDeclareVariable( float3, U,         , );
rtDeclareVariable( float3, V,         , );
rtDeclareVariable( float3, W,         , );

rtBuffer< float4, 2 >        output_buffer;

RT_PROGRAM
void
pinhole_camera( )
{

  size_t2 screenSize = output_buffer.size( );

  float2 d             = make_float2( launch_index ) / make_float2( screenSize ) * 2.f - 1.f;
  float3 ray_origin    = eye;
  float3 ray_direction = normalize( d.x * U + d.y * V + W );

  optix::Ray ray(
                 ray_origin,
                 ray_direction,
                 radiance_ray_type,
                 scene_epsilon
                 );

  PerRayData_radiance prd;
  prd.importance = 1.0f;
  prd.depth      = 0;

  rtTrace( top_object, ray, prd );

  output_buffer[ launch_index ] = make_float4( prd.result, 1.0 );

} // pinhole_camera



//
// Returns solid color for miss rays
//
rtDeclareVariable( float3, bg_color, , );

RT_PROGRAM
void
miss( )
{

  prd_radiance.result = bg_color;

}



//
// Set pixel to solid color upon failure
//
rtDeclareVariable( float3, bad_color, , );

RT_PROGRAM
void
exception( )
{

  output_buffer[ launch_index ] = make_float4( bad_color, 1.0 );

}
