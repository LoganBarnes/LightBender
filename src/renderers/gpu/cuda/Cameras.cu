#include "hip/hip_runtime.h"
#include "optix.h"
#include "RendererObjects.hpp"
#include "path_tracer.h"
#include "random.h"



struct PerRayData_pathtrace
{

  float3 result;
  float3 radiance;
  float3 attenuation;
  float3 origin;
  float3 direction;
  unsigned seed;
  int depth;
  int countEmitted;
  int done;
  int inside;

};


rtDeclareVariable( PerRayData_radiance, prd_radiance,      rtPayload, );

rtDeclareVariable( optix::Ray,          ray,               rtCurrentRay, );
rtDeclareVariable( uint2,               launch_index,      rtLaunchIndex, );
rtDeclareVariable( unsigned int,        frame_number,      , );
rtDeclareVariable( unsigned int,        sqrt_num_samples,  , );

rtDeclareVariable( unsigned int,        radiance_ray_type, , );
rtDeclareVariable( float,               scene_epsilon,     , );
rtDeclareVariable( rtObject,            top_object,        , );


//
// Pinhole camera implementation
//
rtDeclareVariable( float3, eye, , );
rtDeclareVariable( float3, U,   , );
rtDeclareVariable( float3, V,   , );
rtDeclareVariable( float3, W,   , );

rtBuffer< float4, 2 >        output_buffer;

/////////////////////////////////////////////////////////
/// \brief pinhole_camera
/////////////////////////////////////////////////////////
RT_PROGRAM
void
pinhole_camera( )
{

  size_t2 screenSize = output_buffer.size( );

  float2 inv_screen  = 1.0f / make_float2( screenSize ) * 2.0f;
  float2 pixelCorner = make_float2( launch_index ) * inv_screen - 1.0f;

  float2 jitter_scale            = inv_screen / sqrt_num_samples;
  unsigned int samples_per_pixel = sqrt_num_samples * sqrt_num_samples;

  float3 totalRadiance = make_float3( 0.0f );

  // loop vars
  unsigned x, y;
  float2 jitter;

  // faster two for loops for x and y on gpu?
  while ( samples_per_pixel-- )
  {

    x = samples_per_pixel % sqrt_num_samples;
    y = samples_per_pixel / sqrt_num_samples;

    // no random jitter, just center value
    jitter = make_float2( x + 0.5f, y + 0.5f );

    float2 d             = pixelCorner + jitter * jitter_scale;
    float3 ray_origin    = eye;
    float3 ray_direction = normalize( d.x * U + d.y * V + W );

    optix::Ray ray(
                   ray_origin,
                   ray_direction,
                   radiance_ray_type,
                   scene_epsilon
                   );

    PerRayData_radiance prd;
    prd.importance = 1.0f;
    prd.depth      = 0;

    rtTrace( top_object, ray, prd );

    totalRadiance += prd.result;

  }

  totalRadiance /= sqrt_num_samples * sqrt_num_samples ;

  output_buffer[ launch_index ] = make_float4( totalRadiance, 1.0 );

} // pinhole_camera



/////////////////////////////////////////////////////////
/// \brief pinhole_camera
/////////////////////////////////////////////////////////
RT_PROGRAM
void
pathtrace_pinhole_camera( )
{

//  size_t2 screenSize = output_buffer.size( );

//  float2 screenSizeInv = 1.0f / make_float2( screenSize ) * 2.f;
//  float2 pixel         = ( make_float2( launch_index ) ) * screenSizeInv - 1.f;

//  float2 jitter_scale            = screenSizeInv / sqrt_num_samples;
//  unsigned int samples_per_pixel = sqrt_num_samples * sqrt_num_samples;
//  float3 result                  = make_float3( 0.0f );

//  unsigned int seed = tea< 16 >( screenSize.x * launch_index.y + launch_index.x, frame_number );

//  do
//  {

//    unsigned int x       = samples_per_pixel % sqrt_num_samples;
//    unsigned int y       = samples_per_pixel / sqrt_num_samples;
//    float2 jitter        = make_float2( x - rnd( seed ), y - rnd( seed ) );
//    float2 d             = pixel + jitter * jitter_scale;
//    float3 ray_origin    = eye;
//    float3 ray_direction = normalize( d.x * U + d.y * V + W );

//    PerRayData_pathtrace prd;
//    prd.result       = make_float3( 0.f );
//    prd.attenuation  = make_float3( 1.f );
//    prd.countEmitted = true;
//    prd.done         = false;
//    prd.inside       = false;
//    prd.seed         = seed;
//    prd.depth        = 0;

//    for ( ; ; )
//    {

//      Ray ray = make_Ray( ray_origin,
//                         ray_direction,
//                         pathtrace_ray_type,
//                         scene_epsilon,
//                         RT_DEFAULT_MAX );

//      rtTrace( top_object, ray, prd );

//      if ( prd.done )
//      {

//        prd.result += prd.radiance * prd.attenuation;
//        break;

//      }

//      // RR
//      if ( prd.depth >= rr_begin_depth )
//      {

//        float pcont = fmaxf( prd.attenuation );

//        if ( rnd( prd.seed ) >= pcont )
//        {

//          break;

//        }

//        prd.attenuation /= pcont;

//      }

//      prd.depth++;
//      prd.result   += prd.radiance * prd.attenuation;
//      ray_origin    = prd.origin;
//      ray_direction = prd.direction;

//    } // eye ray

//    result += prd.result;
//    seed    = prd.seed;

//  }
//  while ( --samples_per_pixel );

//  float3 pixel_color = result / ( sqrt_num_samples * sqrt_num_samples );

//  if ( frame_number > 1 )
//  {

//    float a          = 1.0f / ( float ) frame_number;
//    float b          = ( ( float ) frame_number - 1.0f ) * a;
//    float3 old_color = make_float3( output_buffer[ launch_index ] );
//    output_buffer[ launch_index ] = make_float4( a * pixel_color + b * old_color, 0.0f );

//  }
//  else
//  {

//    output_buffer[ launch_index ] = make_float4( pixel_color, 0.0f );

//  }

  size_t2 screenSize = output_buffer.size( );

  float2 inv_screen  = 1.0f / make_float2( screenSize ) * 2.0f;
  float2 pixelCorner = make_float2( launch_index ) * inv_screen - 1.0f;

  float2 jitter_scale            = inv_screen / sqrt_num_samples;
  unsigned int samples_per_pixel = sqrt_num_samples * sqrt_num_samples;

  float3 totalRadiance = make_float3( 0.0f );

  // loop vars
  unsigned x, y;
  float2 jitter;

  unsigned seed = tea< 16 >( screenSize.x * launch_index.y + launch_index.x, frame_number );

  // faster two for loops for x and y on gpu?
  while ( samples_per_pixel-- )
  {

    x = samples_per_pixel % sqrt_num_samples;
    y = samples_per_pixel / sqrt_num_samples;

    // no random jitter, just center value
    jitter = make_float2( x + rnd( seed ), y + rnd( seed ) );

    float2 d             = pixelCorner + jitter * jitter_scale;
    float3 ray_origin    = eye;
    float3 ray_direction = normalize( d.x * U + d.y * V + W );

    optix::Ray ray(
                   ray_origin,
                   ray_direction,
                   radiance_ray_type,
                   scene_epsilon
                   );

    PerRayData_radiance prd;
    prd.importance = 1.0f;
    prd.depth      = 0;

    rtTrace( top_object, ray, prd );

    totalRadiance += prd.result;

  }

  totalRadiance /= sqrt_num_samples * sqrt_num_samples ;

  if ( frame_number > 1 )
  {

    float a = 1.0f / static_cast< float >( frame_number );
    float b = ( static_cast< float >( frame_number ) - 1.0f ) * a;
    float3 oldRadiance = make_float3( output_buffer[ launch_index ] );
    output_buffer[ launch_index ] = make_float4( a * totalRadiance + b * oldRadiance, 1.0f );

  }
  else
  {

    output_buffer[ launch_index ] = make_float4( totalRadiance, 1.0f );

  }

} // pinhole_camera



/////////////////////////////////////////////////////////
/// \brief orthographic_camera
/////////////////////////////////////////////////////////
RT_PROGRAM
void
orthographic_camera( )
{

  size_t2 screenSize = output_buffer.size( );

  float2 pixelCorner   = make_float2( launch_index );
  float2 d             = ( pixelCorner + 0.5 ) / make_float2( screenSize ) * 2.f - 1.f; // film coords
  float3 ray_origin    = eye + d.x * U + d.y * V;                          // eye + offset in film space
  float3 ray_direction = normalize( W );                                   // always parallel view direction

  optix::Ray ray = optix::make_Ray(
                                   ray_origin,
                                   ray_direction,
                                   radiance_ray_type,
                                   scene_epsilon,
                                   RT_DEFAULT_MAX
                                   );

  PerRayData_radiance prd;
  prd.importance = 1.f;
  prd.depth      = 0;

  rtTrace( top_object, ray, prd );

  output_buffer[ launch_index ] = make_float4( prd.result, 1.0 );

} // orthographic_camera



/////////////////////////////////////////////////////////
/// \brief pathtrace_orthographic_camera
/////////////////////////////////////////////////////////
RT_PROGRAM
void
pathtrace_orthographic_camera( )
{

  size_t2 screenSize = output_buffer.size( );

  float2 pixelCorner   = make_float2( launch_index );
  float2 d             = ( pixelCorner + 0.5 ) / make_float2( screenSize ) * 2.f - 1.f; // film coords
  float3 ray_origin    = eye + d.x * U + d.y * V;                          // eye + offset in film space
  float3 ray_direction = normalize( W );                                   // always parallel view direction

  optix::Ray ray = optix::make_Ray(
                                   ray_origin,
                                   ray_direction,
                                   radiance_ray_type,
                                   scene_epsilon,
                                   RT_DEFAULT_MAX
                                   );

  PerRayData_radiance prd;
  prd.importance = 1.f;
  prd.depth      = 0;

  rtTrace( top_object, ray, prd );

  float4 radiance = make_float4( prd.result, 1.0 );

  if ( frame_number > 1 )
  {

    float a = 1.0f / static_cast< float >( frame_number );
    float b = ( static_cast< float >( frame_number ) - 1.0f ) * a;
    output_buffer[ launch_index ] = a * radiance + b * output_buffer[ launch_index ];

  }
  else
  {

    output_buffer[ launch_index ] = radiance;

  }

} // orthographic_camera



rtDeclareVariable( float3, bg_color, , );

/////////////////////////////////////////////////////////
/// \brief miss
///
///        Set pixel to solid background color when
///        no itersections are detected
/////////////////////////////////////////////////////////
RT_PROGRAM
void
miss( )
{

  prd_radiance.result = bg_color;

}



rtDeclareVariable( float3, error_color, , );

/////////////////////////////////////////////////////////
/// \brief exception
///
///        Set pixel to solid color upon failure
/////////////////////////////////////////////////////////
RT_PROGRAM
void
exception( )
{

  output_buffer[ launch_index ] = make_float4( error_color, 1.0 );

}
