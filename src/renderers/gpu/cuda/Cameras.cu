#include "hip/hip_runtime.h"
#include "optix.h"
#include "RendererObjects.hpp"
#include "path_tracer.h"
#include "random.h"



rtDeclareVariable( PerRayData_pathtrace, prd_current,       rtPayload, );

rtDeclareVariable( optix::Ray,           ray,               rtCurrentRay,  );
rtDeclareVariable( uint2,                launch_index,      rtLaunchIndex, );

rtDeclareVariable( unsigned int,         frame_number,      , );
rtDeclareVariable( unsigned int,         sqrt_num_samples,  , );

rtDeclareVariable( unsigned int,         radiance_ray_type, , );
rtDeclareVariable( float,                scene_epsilon,     , );
rtDeclareVariable( rtObject,             top_object,        , );

rtDeclareVariable( unsigned int,         max_bounces,       , );
rtDeclareVariable( unsigned int,         first_bounce,      , );
rtDeclareVariable( unsigned int,         globalSeed,        , );


//
// Pinhole camera implementation
//
rtDeclareVariable( float3, eye, , );
rtDeclareVariable( float3, U,   , );
rtDeclareVariable( float3, V,   , );
rtDeclareVariable( float3, W,   , );

rtBuffer< float4, 2 >        output_buffer;

/////////////////////////////////////////////////////////
/// \brief pinhole_camera
/////////////////////////////////////////////////////////
RT_PROGRAM
void
pinhole_camera( )
{

  size_t2 screenSize = output_buffer.size( );

  float2 inv_screen  = 1.0f / make_float2( screenSize ) * 2.0f;
  float2 pixelCorner = make_float2( launch_index ) * inv_screen - 1.0f;

  float2 jitter_scale = inv_screen / sqrt_num_samples;

  float3 totalRadiance = make_float3( 0.0f );

  // loop vars
  unsigned x, y;
  float2 jitter;

  unsigned int samples_per_pixel = sqrt_num_samples * sqrt_num_samples;

  // seems faster than two for loops for x and y on gpu
  while ( samples_per_pixel-- )
  {

    x = samples_per_pixel % sqrt_num_samples;
    y = samples_per_pixel / sqrt_num_samples;

    // no random jitter, just center value
    jitter = make_float2( x + 0.5f, y + 0.5f );

    float2 d             = pixelCorner + jitter * jitter_scale;
    float3 ray_origin    = eye;
    float3 ray_direction = normalize( d.x * U + d.y * V + W );

    PerRayData_pathtrace prd;
    prd.result       = make_float3( 0.f );
    prd.attenuation  = make_float3( 1.f );
    prd.radiance     = make_float3( 0.f );
    prd.countEmitted = true;
    prd.done         = false;
    prd.inside       = false;
    prd.seed         = static_cast< unsigned >( -1 ); // overflow to max value
    prd.depth        = 0;

    optix::Ray ray(
                   ray_origin,
                   ray_direction,
                   radiance_ray_type,
                   scene_epsilon
                   );

    rtTrace( top_object, ray, prd );

    totalRadiance += prd.radiance;

  }

  totalRadiance /= sqrt_num_samples * sqrt_num_samples;

  output_buffer[ launch_index ] = make_float4( totalRadiance, 1.0 );

} // pinhole_camera



/////////////////////////////////////////////////////////
/// \brief pinhole_camera
/////////////////////////////////////////////////////////
RT_PROGRAM
void
pathtrace_pinhole_camera( )
{

  size_t2 screenSize = output_buffer.size( );

  float2 inv_screen  = 1.0f / make_float2( screenSize ) * 2.0f;
  float2 pixelCorner = make_float2( launch_index ) * inv_screen - 1.0f;

  float2 jitter_scale = inv_screen / sqrt_num_samples;

  float3 totalRadiance = make_float3( 0.0f );

  // loop vars
  unsigned x, y;
  float2 jitter;

  unsigned seed = tea< 16 >( screenSize.x * launch_index.y + launch_index.x, frame_number );
  seed += globalSeed;

  unsigned int samples_per_pixel = sqrt_num_samples * sqrt_num_samples;

  // seems faster than two for loops for x and y on gpu
  while ( samples_per_pixel-- )
  {

    x = samples_per_pixel % sqrt_num_samples;
    y = samples_per_pixel / sqrt_num_samples;

    // random jitter within sample area
    jitter = make_float2( x + rnd( seed ), y + rnd( seed ) );

    float2 d             = pixelCorner + jitter * jitter_scale;
    float3 ray_origin    = eye;
    float3 ray_direction = normalize( d.x * U + d.y * V + W );

    PerRayData_pathtrace prd;
    prd.result       = make_float3( 0.f );
    prd.attenuation  = make_float3( 1.f );
    prd.radiance     = make_float3( 0.f );
    prd.countEmitted = true;
    prd.done         = false;
    prd.inside       = false;
    prd.seed         = seed;
    prd.depth        = 0;

    for ( ; ; )
    {

      float3 attenuation = prd.attenuation;

      optix::Ray ray(
                     ray_origin,
                     ray_direction,
                     radiance_ray_type,
                     scene_epsilon
                     );

      rtTrace( top_object, ray, prd );


      if ( prd.depth >= max_bounces )
      {

        prd.result += prd.radiance * attenuation;
        break;

      }

      if ( prd.depth >= first_bounce )
      {

        prd.result += prd.radiance * attenuation;

      }

      if ( prd.done )
      {

        break;

      }

      ++prd.depth;
      ray_origin    = prd.origin;
      ray_direction = prd.direction;

    }

    totalRadiance += prd.result;

  }

  totalRadiance /= sqrt_num_samples * sqrt_num_samples;

  if ( frame_number > 1 )
  {

    float a            = 1.0f / static_cast< float >( frame_number );
    float b            = ( static_cast< float >( frame_number ) - 1.0f ) * a;
    float3 oldRadiance = make_float3( output_buffer[ launch_index ] );
    output_buffer[ launch_index ] = make_float4( a * totalRadiance + b * oldRadiance, 1.0f );

  }
  else
  {

    output_buffer[ launch_index ] = make_float4( totalRadiance, 1.0f );

  }

} // pinhole_camera



/////////////////////////////////////////////////////////
/// \brief orthographic_camera
/////////////////////////////////////////////////////////
RT_PROGRAM
void
orthographic_camera( )
{

  size_t2 screenSize = output_buffer.size( );

  float2 inv_screen  = 1.0f / make_float2( screenSize ) * 2.0f;
  float2 pixelCorner = make_float2( launch_index ) * inv_screen - 1.0f;

  float2 jitter_scale = inv_screen / sqrt_num_samples;

  float3 totalRadiance = make_float3( 0.0f );

  // loop vars
  unsigned x, y;
  float2 jitter;

  unsigned int samples_per_pixel = sqrt_num_samples * sqrt_num_samples;

  // seems faster than two for loops for x and y on gpu
  while ( samples_per_pixel-- )
  {

    x = samples_per_pixel % sqrt_num_samples;
    y = samples_per_pixel / sqrt_num_samples;

    // no random jitter, just center value
    jitter = make_float2( x + 0.5f, y + 0.5f );

    float2 d             = pixelCorner + jitter * jitter_scale;
    float3 ray_origin    = eye + d.x * U + d.y * V; // eye + offset in film space
    float3 ray_direction = normalize( W );          // always parallel view direction

    PerRayData_pathtrace prd;
    prd.result       = make_float3( 0.f );
    prd.attenuation  = make_float3( 1.f );
    prd.radiance     = make_float3( 0.f );
    prd.countEmitted = true;
    prd.done         = false;
    prd.inside       = false;
    prd.seed         = static_cast< unsigned >( -1 ); // overflow to max value
    prd.depth        = 0;

    optix::Ray ray(
                   ray_origin,
                   ray_direction,
                   radiance_ray_type,
                   scene_epsilon
                   );

    rtTrace( top_object, ray, prd );

    totalRadiance += prd.radiance;

  }

  totalRadiance /= sqrt_num_samples * sqrt_num_samples;

  output_buffer[ launch_index ] = make_float4( totalRadiance, 1.0 );

} // orthographic_camera



/////////////////////////////////////////////////////////
/// \brief pathtrace_orthographic_camera
/////////////////////////////////////////////////////////
RT_PROGRAM
void
pathtrace_orthographic_camera( )
{

  size_t2 screenSize = output_buffer.size( );

  float2 inv_screen  = 1.0f / make_float2( screenSize ) * 2.0f;
  float2 pixelCorner = make_float2( launch_index ) * inv_screen - 1.0f;

  float2 jitter_scale = inv_screen / sqrt_num_samples;

  float3 totalRadiance = make_float3( 0.0f );

  // loop vars
  unsigned x, y;
  float2 jitter;

  unsigned seed = tea< 16 >( screenSize.x * launch_index.y + launch_index.x, frame_number );
  seed += globalSeed;

  unsigned int samples_per_pixel = sqrt_num_samples * sqrt_num_samples;

  // seems faster than two for loops for x and y on gpu
  while ( samples_per_pixel-- )
  {

    x = samples_per_pixel % sqrt_num_samples;
    y = samples_per_pixel / sqrt_num_samples;

    // random jitter within sample area
    jitter = make_float2( x + rnd( seed ), y + rnd( seed ) );

    float2 d             = pixelCorner + jitter * jitter_scale;
    float3 ray_origin    = eye + d.x * U + d.y * V; // eye + offset in film space
    float3 ray_direction = normalize( W );          // always parallel view direction


    PerRayData_pathtrace prd;
    prd.result       = make_float3( 0.f );
    prd.attenuation  = make_float3( 1.f );
    prd.radiance     = make_float3( 0.f );
    prd.countEmitted = true;
    prd.done         = false;
    prd.inside       = false;
    prd.seed         = seed;
    prd.depth        = 0;

    for ( ; ; )
    {

      float3 attenuation = prd.attenuation;

      optix::Ray ray(
                     ray_origin,
                     ray_direction,
                     radiance_ray_type,
                     scene_epsilon
                     );

      rtTrace( top_object, ray, prd );


      if ( prd.depth > max_bounces )
      {

        prd.result += prd.radiance * attenuation;
        break;

      }

      if ( prd.depth >= first_bounce )
      {

        prd.result += prd.radiance * attenuation;

      }

      if ( prd.done )
      {

        break;

      }

      ++prd.depth;
      ray_origin    = prd.origin;
      ray_direction = prd.direction;

    }

    totalRadiance += prd.result;

  }

  totalRadiance /= sqrt_num_samples * sqrt_num_samples;

  if ( frame_number > 1 )
  {

    float a            = 1.0f / static_cast< float >( frame_number );
    float b            = ( static_cast< float >( frame_number ) - 1.0f ) * a;
    float3 oldRadiance = make_float3( output_buffer[ launch_index ] );
    output_buffer[ launch_index ] = make_float4( a * totalRadiance + b * oldRadiance, 1.0f );

  }
  else
  {

    output_buffer[ launch_index ] = make_float4( totalRadiance, 1.0f );

  }

} // orthographic_camera



rtDeclareVariable( float3, bg_color, , );

/////////////////////////////////////////////////////////
/// \brief miss
///
///        Set pixel to solid background color when
///        no itersections are detected
/////////////////////////////////////////////////////////
RT_PROGRAM
void
miss( )
{

  prd_current.radiance = bg_color;
  prd_current.done     = true;

}



rtDeclareVariable( float3, error_color, , );

/////////////////////////////////////////////////////////
/// \brief exception
///
///        Set pixel to solid color upon failure
/////////////////////////////////////////////////////////
RT_PROGRAM
void
exception( )
{

  output_buffer[ launch_index ] = make_float4( error_color, 1.0 );

}
