#include "hip/hip_runtime.h"
#include "optix.h"
#include "RendererObjects.hpp"

rtDeclareVariable( PerRayData_radiance, prd_radiance,      rtPayload, );

rtDeclareVariable( optix::Ray,          ray,               rtCurrentRay, );
rtDeclareVariable( uint2,               launch_index,      rtLaunchIndex, );

rtDeclareVariable( unsigned int,        radiance_ray_type, , );
rtDeclareVariable( float,               scene_epsilon,     , );
rtDeclareVariable( rtObject,            top_object,        , );


//
// Pinhole camera implementation
//
rtDeclareVariable( float3, eye, , );
rtDeclareVariable( float3, U,   , );
rtDeclareVariable( float3, V,   , );
rtDeclareVariable( float3, W,   , );

rtBuffer< float4, 2 >        output_buffer;


/////////////////////////////////////////////////////////
/// \brief pinhole_camera
/////////////////////////////////////////////////////////
RT_PROGRAM
void
pinhole_camera( )
{

  size_t2 screenSize = output_buffer.size( );

  float2 d             = make_float2( launch_index ) / make_float2( screenSize ) * 2.f - 1.f;
  float3 ray_origin    = eye;
  float3 ray_direction = normalize( d.x * U + d.y * V + W );

  optix::Ray ray(
                 ray_origin,
                 ray_direction,
                 radiance_ray_type,
                 scene_epsilon
                 );

  PerRayData_radiance prd;
  prd.importance = 1.0f;
  prd.depth      = 0;

  rtTrace( top_object, ray, prd );

  output_buffer[ launch_index ] = make_float4( prd.result, 1.0 );

} // pinhole_camera



/////////////////////////////////////////////////////////
/// \brief orthographic_camera
/////////////////////////////////////////////////////////
RT_PROGRAM
void
orthographic_camera( )
{

  size_t2 screen = output_buffer.size( );

  float2 d             = make_float2( launch_index ) / make_float2( screen ) * 2.f - 1.f; // film coords
  float3 ray_origin    = eye + d.x * U + d.y * V;                          // eye + offset in film space
  float3 ray_direction = normalize( W );                                   // always parallel view direction

  optix::Ray ray = optix::make_Ray(
                                   ray_origin,
                                   ray_direction,
                                   radiance_ray_type,
                                   scene_epsilon,
                                   RT_DEFAULT_MAX
                                   );

  PerRayData_radiance prd;
  prd.importance = 1.f;
  prd.depth      = 0;

  rtTrace( top_object, ray, prd );

  output_buffer[ launch_index ] = make_float4( prd.result, 1.0 );

} // orthographic_camera



rtDeclareVariable( float3, bg_color, , );

/////////////////////////////////////////////////////////
/// \brief miss
///
///        Set pixel to solid background color when
///        no itersections are detected
/////////////////////////////////////////////////////////
RT_PROGRAM
void
miss( )
{

  prd_radiance.result = bg_color;

}



rtDeclareVariable( float3, error_color, , );

/////////////////////////////////////////////////////////
/// \brief exception
///
///        Set pixel to solid color upon failure
/////////////////////////////////////////////////////////
RT_PROGRAM
void
exception( )
{

  output_buffer[ launch_index ] = make_float4( error_color, 1.0 );

}
