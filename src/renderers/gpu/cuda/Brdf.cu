#include "hip/hip_runtime.h"
#include "optix.h"
#include "RendererObjects.hpp"
#include "commonStructs.h"


rtDeclareVariable( float3,              shading_normal,   attribute shading_normal, );
rtDeclareVariable( float3,              geometric_normal, attribute geometric_normal, );

rtDeclareVariable( PerRayData_radiance, prd_radiance,     rtPayload, );
rtDeclareVariable( PerRayData_shadow,   prd_shadow,       rtPayload, );

rtDeclareVariable( optix::Ray,          ray,              rtCurrentRay, );
rtDeclareVariable( float,               t_hit,            rtIntersectionDistance, );


rtDeclareVariable( unsigned int,        shadow_ray_type,  , );
rtDeclareVariable( float,               scene_epsilon,    , );
rtDeclareVariable( rtObject,            top_object,       , );

rtBuffer< BasicLight > lights;



/////////////////////////////////////////////////////////
/// \brief closest_hit_normals
///
///        Sets shading normal as the surface color
/////////////////////////////////////////////////////////
RT_PROGRAM
void
closest_hit_normals( )
{

  float3 worldGeoNormal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 worldShadeNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 ffnormal         = faceforward( worldShadeNormal, -ray.direction, worldGeoNormal );

  prd_radiance.result = ffnormal * 0.5f + 0.5f;

}



/////////////////////////////////////////////////////////
/// \brief closest_hit_simple_shading
/////////////////////////////////////////////////////////
RT_PROGRAM
void
closest_hit_simple_shading( )
{

  float3 worldGeoNormal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 worldShadeNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 ffnormal         = faceforward( worldShadeNormal, -ray.direction, worldGeoNormal );

  float3 radiance = make_float3( 0.0f );

  float3 hit_point = ray.origin + t_hit * ray.direction;

  for ( int i = 0; i < lights.size( ); ++i )
  {

    BasicLight &light = lights[ i ];

    // direction and distance to light
    float3 w_i            = light.pos - hit_point;
    float distToLightPow2 = dot( w_i, w_i );
    float distToLight     = sqrt( distToLightPow2 );
    w_i /= distToLight; // normalizes w_i

    float cosAngle = dot( ffnormal, w_i );

    if ( cosAngle > 0.0f )
    {

      // results from shadow ray
      PerRayData_shadow shadow_prd;
      shadow_prd.attenuation = optix::make_float3( 1.0f );

      // shadow ray
      optix::Ray shadow_ray(
                            hit_point,
                            w_i,
                            shadow_ray_type,
                            scene_epsilon,
                            distToLight
                            );

      // shoot ray into scene
      rtTrace( top_object, shadow_ray, shadow_prd );


      radiance += ( 1.0 / M_PIf )                        // lambertian pi normalization
                  * ( light.radiance / distToLightPow2 ) // incident radiance
                  * cosAngle                             // angle between normal and incident ray
                  * shadow_prd.attenuation;              // attenuation from shadowing objects

    }

  }

  prd_radiance.result = radiance;

} // closest_hit_simple_shading



//rtDeclareVariable( float3, albedo,    , );
//rtDeclareVariable( float,  roughness, , );

/////////////////////////////////////////////////////////
/// \brief closest_hit_bsdf
/////////////////////////////////////////////////////////
RT_PROGRAM
void
closest_hit_bsdf( )
{

  float3 albedo   = make_float3( 0.71f, 0.62f, 0.53f );
  float roughness = 0.3f;

  float3 worldGeoNormal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 worldShadeNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 ffnormal         = faceforward( worldShadeNormal, -ray.direction, worldGeoNormal );

  float3 radiance = make_float3( 0.0f );

  float3 hit_point = ray.origin + t_hit * ray.direction;

  float3 w_o = -ray.direction;

  for ( int i = 0; i < lights.size( ); ++i )
  {

    BasicLight &light = lights[ i ];

    // direction and distance to light
    float3 w_i            = light.pos - hit_point;
    float distToLightPow2 = dot( w_i, w_i );
    float distToLight     = sqrt( distToLightPow2 );
    w_i /= distToLight; // normalizes w_i

    float cosAngle = dot( ffnormal, w_i );

    float3 localRadiance;

    if ( cosAngle > 0.0f )
    {

      // results from shadow ray
      PerRayData_shadow shadow_prd;
      shadow_prd.attenuation = optix::make_float3( 1.0f );

      // shadow ray
      optix::Ray shadow_ray(
                            hit_point,
                            w_i,
                            shadow_ray_type,
                            scene_epsilon,
                            distToLight
                            );

      // shoot ray into scene
      rtTrace( top_object, shadow_ray, shadow_prd );


      // only missing bsdf calculation right now
      localRadiance = ( light.radiance / distToLightPow2 ) // incident radiance
                      * cosAngle                           // angle between normal and incident ray
                      * shadow_prd.attenuation;            // attenuation from shadowing objects

      if ( dot( localRadiance, localRadiance ) > 1.0e-9f )
      {

        // brdf calculation

        // oren nayar diffuse brdf
        float gammaPow2 = roughness * roughness;

        float nDotL = optix::dot( ffnormal, w_i );
        float nDotV = optix::dot( ffnormal, w_o );

        float s = optix::dot( w_i, w_o ) - nDotL * nDotV;

        float t = s <= 0.0f ? 1.0f : max( nDotL, nDotV );

        float3 A = ( 1.0
                    - 0.5  * ( gammaPow2 / ( gammaPow2 + 0.33 ) )
                    + 0.17 * ( gammaPow2 / ( gammaPow2 + 0.13 ) ) * albedo
                    ) / M_PIf;

        float B = 0.45f * ( gammaPow2 / ( gammaPow2 + 0.09f ) ) / M_PIf;

        float3 onBrdf = albedo * ( A + B * s / t );

        radiance += localRadiance * onBrdf;

      }

    }

  }

  prd_radiance.result = radiance;

} // closest_hit_bsdf



/////////////////////////////////////////////////////////
/// \brief any_hit_occlusion
/////////////////////////////////////////////////////////
RT_PROGRAM
void
any_hit_occlusion( )
{

  // always opaque for now
  prd_shadow.attenuation = optix::make_float3( 0.0f );

  rtTerminateRay( );

}
