#include "hip/hip_runtime.h"
#include "optix.h"
#include "RendererObjects.hpp"
#include "commonStructs.h"


rtDeclareVariable( float3, shading_normal,   attribute shading_normal, );
rtDeclareVariable( float3, geometric_normal, attribute geometric_normal, );

rtDeclareVariable( PerRayData_radiance, prd_radiance, rtPayload, );


rtDeclareVariable( optix::Ray, ray,   rtCurrentRay, );
rtDeclareVariable( float,      t_hit, rtIntersectionDistance, );

rtBuffer< BasicLight > lights;


//
// Returns shading normal as the surface shading result
//
RT_PROGRAM
void
closest_hit_normals( )
{

  float3 world_geo_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 world_shade_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 ffnormal           = faceforward( world_shade_normal, -ray.direction, world_geo_normal );

  prd_radiance.result = ffnormal * 0.5f + 0.5f;

}



RT_PROGRAM
void
closest_hit_simple_shading( )
{

  float3 world_geo_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 world_shade_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 ffnormal           = faceforward( world_shade_normal, -ray.direction, world_geo_normal );

  float3 radiance = make_float3( 0.0f );

  float3 hit_point = ray.origin + t_hit * ray.direction;

  for( int i = 0; i < lights.size( ); ++i )
  {

    BasicLight &light = lights[ i ];

    // direction and distance to light
    float3 w_i             = light.pos - hit_point;
    float  distToLightSqrd = dot( w_i, w_i );
    w_i                   /= sqrt( distToLightSqrd );

    float  cosAngle = max( 0.0, dot( ffnormal, w_i ) );

    radiance += cosAngle * light.radiance / distToLightSqrd;

  }

  prd_radiance.result = radiance;

}
