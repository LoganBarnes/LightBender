#include "hip/hip_runtime.h"
#include "optix.h"
#include "RendererObjects.hpp"
#include "commonStructs.h"


rtDeclareVariable( float3,              shading_normal,   attribute shading_normal, );
rtDeclareVariable( float3,              geometric_normal, attribute geometric_normal, );

rtDeclareVariable( PerRayData_radiance, prd_radiance,     rtPayload, );
rtDeclareVariable( PerRayData_shadow,   prd_shadow,       rtPayload, );

rtDeclareVariable( optix::Ray,          ray,              rtCurrentRay, );
rtDeclareVariable( float,               t_hit,            rtIntersectionDistance, );


rtDeclareVariable( unsigned int,        shadow_ray_type,  , );
rtDeclareVariable( float,               scene_epsilon,    , );
rtDeclareVariable( rtObject,            top_shadower,     , );

rtBuffer< BasicLight > lights;



/////////////////////////////////////////////////////////
/// \brief closest_hit_normals
///
///        Sets shading normal as the surface color
/////////////////////////////////////////////////////////
RT_PROGRAM
void
closest_hit_normals( )
{

  float3 worldGeoNormal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 worldShadeNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 ffnormal         = faceforward( worldShadeNormal, -ray.direction, worldGeoNormal );

  prd_radiance.result = ffnormal * 0.5f + 0.5f;

}



/////////////////////////////////////////////////////////
/// \brief closest_hit_simple_shading
/////////////////////////////////////////////////////////
RT_PROGRAM
void
closest_hit_simple_shading( )
{

  float3 worldGeoNormal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 worldShadeNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 ffnormal         = faceforward( worldShadeNormal, -ray.direction, worldGeoNormal );

  float3 radiance = make_float3( 0.0f );

  float3 hit_point = ray.origin + t_hit * ray.direction;

  for ( int i = 0; i < lights.size( ); ++i )
  {

    BasicLight &light = lights[ i ];

    // direction and distance to light
    float3 w_i            = light.pos - hit_point;
    float distToLightPow2 = dot( w_i, w_i );
    float distToLight     = sqrt( distToLightPow2 );
    w_i /= distToLight; // normalizes w_i

    float cosAngle = dot( ffnormal, w_i );

    if ( cosAngle > 0.0f )
    {

      // results from shadow ray
      PerRayData_shadow shadow_prd;
      shadow_prd.attenuation = optix::make_float3( 1.0f );

      // shadow ray
      optix::Ray shadow_ray(
                            hit_point,
                            w_i,
                            shadow_ray_type,
                            scene_epsilon,
                            distToLight
                            );

      // shoot ray into scene
      rtTrace( top_shadower, shadow_ray, shadow_prd );


      // only missing bsdf calculation right now
      radiance += ( light.radiance / distToLightPow2 ) // incident radiance
                  * cosAngle                           // angle between normal and incident ray
                  * shadow_prd.attenuation;            // attenuation from shadowing objects

    }

  }

  prd_radiance.result = radiance;

} // closest_hit_simple_shading



/////////////////////////////////////////////////////////
/// \brief any_hit_occlusion
/////////////////////////////////////////////////////////
RT_PROGRAM
void
any_hit_occlusion( )
{

  // always opaque for now
  prd_shadow.attenuation = optix::make_float3( 0.0f );

  rtTerminateRay( );

}
