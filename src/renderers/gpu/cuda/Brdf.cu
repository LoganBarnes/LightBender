#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_stream_namespace.h>
#include "commonStructs.h"
#include "random.h"
#include "RendererObjects.hpp" // should be last to avoid FLT_MAX redefintion warning



//////////////////////////////////////////////////////////////
/// \brief createONB
///
///        Create Orthonormal Basis from normalized vector
//////////////////////////////////////////////////////////////
static
__device__ __inline__
void
createONB(
          const float3 &n, ///< normal
          float3       &U, ///< output U vector
          float3       &V  ///< output V vector
          )
{

  U = cross( n, make_float3( 0.0f, 1.0f, 0.0f ) );

  if ( dot( U, U ) < 1.e-3f )
  {

    U = cross( n, make_float3( 1.0f, 0.0f, 0.0f ) );

  }

  U = normalize( U );
  V = cross( n, U );

}



//////////////////////////////////////////////////////////////
/// \brief sampleLight
///
///        Choose a random point from a spherical light
///
/// \return
//////////////////////////////////////////////////////////////
static
__device__ __inline__
float3
sampleIlluminator(
                  unsigned             &seed,        ///< random seed
                  const SurfaceElement &surfel,      ///< info about the current surface
                  const Illuminator    &illuminator, ///< info about the curren illuminator
                  float               *pPdf          ///< output pdf value
                  )
{

  float theta = rnd( seed ) * 2 * M_PIf;
  float u     = rnd( seed ) * 2.0 - 1.0;

  float xyCoeff = sqrt( 1.0 - u * u );

  float3 samplePos = make_float3(
                                 xyCoeff * cos( theta ),
                                 xyCoeff * sin( theta ),
                                 u
                                 );

  // sample on hemisphere in direction of point
  if ( dot( samplePos, normalize( surfel.point - illuminator.center ) ) < 0.0f )
  {

    samplePos = -samplePos;

  }

  *pPdf = M_PIf;

  samplePos = illuminator.center + samplePos * illuminator.radius;

  return samplePos;

} // sampleIlluminator



static
__device__ __inline__
float3
calculateSpecular(
                  const float3         &V,
                  const float3         &L,
                  const float3         &F,
                  const SurfaceElement &surfel
                  )
{

  // roughness -> 'm' in cook-torrance lingo
  float m = surfel.material.roughness;

  float3 H = normalize( V + L );

  float cosNV = dot( surfel.normal, V );
  float cosNH = dot( surfel.normal, H );
  float cosNL = dot( surfel.normal, L );
  float cosVH = dot( V, H );

  // geometric attenuation
  float G = min( 1.0f, min( 2.0f * cosNH * cosNV / cosVH, 2.0f * cosNH * cosNL / cosVH ) );

  // microfacet slope distribution
  float cosNHPow2 = cosNH * cosNH;
  float mPo2      = m * m;

  float D = ( 1.0 / ( M_PIf * mPo2 * cosNHPow2 * cosNHPow2 ) )
            * exp( ( cosNHPow2 - 1.0 ) / ( mPo2 * cosNHPow2 ) );

  // return mat.albedo * ( F * D * G ) / ( PI * cosNL * cosNV );
  // return mat.albedo * F;
  float3 specular = surfel.material.albedo * ( F * D * G ) / ( M_PIf * cosNL * cosNV );

  float3 diffuse = surfel.material.albedo * ( 1.0 - F ) / M_PIf;

  return diffuse + specular;

} // calculateSpecular



rtDeclareVariable( float3,               shading_normal,   attribute shading_normal, );
rtDeclareVariable( float3,               geometric_normal, attribute geometric_normal, );

rtDeclareVariable( PerRayData_pathtrace, prd_current,      rtPayload, );
rtDeclareVariable( PerRayData_shadow,    prd_shadow,       rtPayload, );

rtDeclareVariable( optix::Ray,           ray,              rtCurrentRay, );
rtDeclareVariable( float,                t_hit,            rtIntersectionDistance, );


rtDeclareVariable( unsigned int,         shadow_ray_type,  , );
rtDeclareVariable( float,                scene_epsilon,    , );
rtDeclareVariable( rtObject,             top_shadower,     , );

rtBuffer< Illuminator > illuminators;



/////////////////////////////////////////////////////////
/// \brief closest_hit_normals
///
///        Sets shading normal as the surface color
/////////////////////////////////////////////////////////
RT_PROGRAM
void
closest_hit_normals( )
{

  float3 worldGeoNormal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 worldShadeNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 ffnormal         = faceforward( worldShadeNormal, -ray.direction, worldGeoNormal );

  prd_current.radiance = ffnormal * 0.5f + 0.5f;
  prd_current.done     = true;

}



/////////////////////////////////////////////////////////
/// \brief closest_hit_simple_shading
/////////////////////////////////////////////////////////
RT_PROGRAM
void
closest_hit_simple_shading( )
{

  const float3 simpleShadeAlbedo = make_float3( 0.8f );

  SurfaceElement surfel;

  float3 worldGeoNormal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 worldShadeNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );

  surfel.normal = faceforward( worldShadeNormal, -ray.direction, worldGeoNormal );

  float3 radiance = make_float3( 0.0f );

  surfel.point = ray.origin + t_hit * ray.direction;


  // loop vars
  float3 w_i;
  float distToLightPow2, distToLight;

  for ( int i = 0; i < illuminators.size( ); ++i )
  {

    Illuminator &illuminator = illuminators[ i ];

    float3 lightPos = illuminator.center;
    float3 flux     = illuminator.radiantFlux;

    float totalDistPow2;
    float pdf        = M_PIf;
    float mis_weight = 1.0f;

    // randomly sample sphere (only light shape for now)
    if ( prd_current.seed != static_cast< unsigned >( -1 ) )
    {

      lightPos = sampleIlluminator( prd_current.seed, surfel, illuminator, &pdf );
//      mis_weight = 0.5f;

      // direction and distance to light
      w_i             = lightPos - surfel.point;
      distToLightPow2 = dot( w_i, w_i );
      distToLight     = sqrt( distToLightPow2 );
      w_i            /= distToLight; // normalizes w_i

      totalDistPow2  = distToLight + illuminator.radius;
      totalDistPow2 *= totalDistPow2;

      // lambertian emitter
      ///\todo: Sample by sollid angle for quicker convergance
      flux *= 0.5f * max( 0.0, dot( -w_i, normalize( lightPos - illuminator.center ) ) );

    }
    else
    {

      // direction and distance to light
      w_i             = lightPos - surfel.point;
      distToLightPow2 = dot( w_i, w_i );
      distToLight     = sqrt( distToLightPow2 );
      w_i            /= distToLight; // normalizes w_i

      totalDistPow2  = distToLight;
      totalDistPow2 *= totalDistPow2;

      flux /= 4.0f;

    }


    float cosAngle = dot( surfel.normal, w_i );

    if ( cosAngle > 0.0f )
    {

      // results from shadow ray
      PerRayData_shadow shadow_prd;
      shadow_prd.attenuation = optix::make_float3( 1.0f );

      // shadow ray
      optix::Ray shadow_ray(
                            surfel.point,
                            w_i,
                            shadow_ray_type,
                            scene_epsilon,
                            distToLight
                            );

      // shoot ray into scene
      rtTrace( top_shadower, shadow_ray, shadow_prd );


      radiance += ( simpleShadeAlbedo / M_PIf ) // lambertian pi normalization
                  * ( flux / totalDistPow2 )    // inverse square law
                  * ( mis_weight / pdf )        // importance weight
                  * cosAngle                    // angle between normal and incident ray
                  * shadow_prd.attenuation;     // attenuation from shadowing objects

    }

  }


  //
  // next ray for indirect light
  //
  if ( prd_current.seed != static_cast< unsigned >( -1 ) )
  {

    float scatterProb = ( simpleShadeAlbedo.x + simpleShadeAlbedo.y + simpleShadeAlbedo.z ) / 3;

    float rouletteVal = rnd( prd_current.seed );

    //
    // scatter
    //
    rouletteVal -= scatterProb;

    if ( rouletteVal <= 0.0f )
    {

      prd_current.origin = surfel.point;

      float z1 = rnd( prd_current.seed );
      float z2 = rnd( prd_current.seed );
      float3 p;

      optix::cosine_sample_hemisphere( z1, z2, p );

      float3 v1, v2;
      createONB( surfel.normal, v1, v2 );

      prd_current.direction    = v1 * p.x + v2 * p.y + surfel.normal * p.z;
      prd_current.attenuation *= simpleShadeAlbedo / scatterProb;
      prd_current.countEmitted = false;

      prd_current.radiance = radiance;
      return;

    }

    //
    // absorb
    //
    prd_current.done = true;

  }

  prd_current.radiance = radiance;

} // closest_hit_simple_shading



// albedo:
// 0.13f // moon
// 0.71f, 0.62f, 0.53f // clay
// roughness = 0.3f;

rtDeclareVariable( float3, albedo,    , );
rtDeclareVariable( float,  roughness, , );
rtDeclareVariable( float3, ior,       , );

/////////////////////////////////////////////////////////
/// \brief closest_hit_bsdf
/////////////////////////////////////////////////////////
RT_PROGRAM
void
closest_hit_bsdf( )
{

//  float k = 0.5;

  SurfaceElement surfel;

  surfel.material.albedo    = albedo;
  surfel.material.roughness = roughness;
  surfel.material.IOR       = ior;

  float3 worldGeoNormal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 worldShadeNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );

  surfel.normal = faceforward( worldShadeNormal, -ray.direction, worldGeoNormal );

  float3 radiance = make_float3( 0.0f );

  surfel.point = ray.origin + t_hit * ray.direction;

  // view vector
  float3 w_v = -ray.direction;


  //
  // fresnel calculation for current surface
  //
  float3 currentIOR = make_float3( 1.0 ); // air (no transmission yet)

  float cosNV = dot( surfel.normal, w_v );

  float3 eta = currentIOR / surfel.material.IOR;
  float3 cosT;

  // individual fresnel calc for each RGB wavelength
  float3 T = refract( -w_v, surfel.normal, eta.x );
  cosT.x = dot( -surfel.normal, T );

  T      = refract( -w_v, surfel.normal, eta.y );
  cosT.y = dot( -surfel.normal, T );

  T      = refract( -w_v, surfel.normal, eta.z );
  cosT.z = dot( -surfel.normal, T );

  float3 F = fresnel( make_float3( cosNV ), cosT, currentIOR, surfel.material.IOR );



  // loop vars
  float3 w_l; // light vector
  float distToLightPow2, distToLight;

  for ( int i = 0; i < illuminators.size( ); ++i )
  {

    Illuminator &illuminator = illuminators[ i ];

    float3 lightPos = illuminator.center;
    float3 flux     = illuminator.radiantFlux;

    float totalDistPow2;
    float pdf        = M_PIf;
    float mis_weight = 1.0f;

    // randomly sample sphere (only light shape for now)
    if ( prd_current.seed != static_cast< unsigned >( -1 ) )
    {

      lightPos = sampleIlluminator( prd_current.seed, surfel, illuminator, &pdf );
//      mis_weight = 0.5;

      // direction and distance to light
      w_l             = lightPos - surfel.point;
      distToLightPow2 = dot( w_l, w_l );
      distToLight     = sqrt( distToLightPow2 );
      w_l            /= distToLight; // normalizes w_i

      totalDistPow2  = distToLight + illuminator.radius;
      totalDistPow2 *= totalDistPow2;

      // lambertian emitter
      ///\todo: Sample by sollid angle for quicker convergance
      flux *= 0.5f * max( 0.0, dot( -w_l, normalize( lightPos - illuminator.center ) ) );

    }
    else
    {

      // direction and distance to light
      w_l             = lightPos - surfel.point;
      distToLightPow2 = dot( w_l, w_l );
      distToLight     = sqrt( distToLightPow2 );
      w_l            /= distToLight; // normalizes w_i

      totalDistPow2  = distToLight;
      totalDistPow2 *= totalDistPow2;

      flux /= 4.0f;

    }


    float cosNL = dot( surfel.normal, w_l );

    float3 localRadiance;

    if ( cosNL > 0.0f )
    {

      // results from shadow ray
      PerRayData_shadow shadow_prd;
      shadow_prd.attenuation = optix::make_float3( 1.0f );

      // shadow ray
      optix::Ray shadow_ray(
                            surfel.point,
                            w_l,
                            shadow_ray_type,
                            scene_epsilon,
                            distToLight
                            );

      // shoot ray into scene
      rtTrace( top_shadower, shadow_ray, shadow_prd );


      // bsdf calculation added below
      localRadiance = ( flux / totalDistPow2 )  // incident radiance
                      * ( mis_weight / pdf )    // importance weighting
                      * cosNL                   // angle between normal and incident ray
                      * shadow_prd.attenuation; // attenuation from shadowing objects


      if ( dot( localRadiance, localRadiance ) > 1.0e-9f )
      {

        //
        // bsdf calculation
        //

        //
        // cook-torrance specular
        //
        float3 specular = make_float3( 0.0f );

        if ( prd_current.useSpecular )
        {

          specular = calculateSpecular( w_v, w_l, F, surfel );

        }

        //
        // oren nayar diffuse brdf
        //
        float gammaPow2 = surfel.material.roughness * surfel.material.roughness;

        float nDotL = optix::dot( surfel.normal, w_l );
        float nDotV = optix::dot( surfel.normal, w_v );

        float s = optix::dot( w_l, w_v ) - nDotL * nDotV;

        float t = s <= 0.0f ? 1.0f : max( nDotL, nDotV );

        float3 A = ( 1.0
                    - 0.5  * ( gammaPow2 / ( gammaPow2 + 0.33 ) )
                    + 0.17 * ( gammaPow2 / ( gammaPow2 + 0.13 ) ) * surfel.material.albedo
                    ) / M_PIf;

        float B = 0.45f * ( gammaPow2 / ( gammaPow2 + 0.09f ) ) / M_PIf;

        float3 diffuse = surfel.material.albedo * ( A + B * s / t );

        radiance += localRadiance * ( diffuse * ( 1.0f - F ) + specular );

      }

    }

  }


  //
  // next ray for indirect light
  //
  if ( prd_current.seed != static_cast< unsigned >( -1 ) )
  {

    float reflectProb = ( F.x + F.y + F.z ) / 3;

    float scatterProb = ( albedo.x + albedo.y + albedo.z ) / 3;
//    scatterProb *= 1.0 - reflectProb;

    //
    // russian roulette based on scattering probabilities
    //
    float rouletteVal = rnd( prd_current.seed );

    //
    // diffuse scatter
    //
    rouletteVal -= scatterProb;

    if ( rouletteVal <= 0.0f )
    {

      prd_current.origin = surfel.point;

      float z1 = rnd( prd_current.seed );
      float z2 = rnd( prd_current.seed );
      float3 p;

      optix::cosine_sample_hemisphere( z1, z2, p );

      float3 v1, v2;
      createONB( surfel.normal, v1, v2 );

      prd_current.direction    = v1 * p.x + v2 * p.y + surfel.normal * p.z;
      prd_current.attenuation *= albedo / scatterProb;  // use the albedo as the diffuse response
      prd_current.countEmitted = false;
      prd_current.useSpecular  = false;

      prd_current.radiance = radiance;
      return;

    }


    //
    // reflect
    //
    rouletteVal -= reflectProb;

    if ( rouletteVal <= 0.0f )
    {

      //
      // sample from raised cosine distribution
      //
      float z1 = rnd( prd_current.seed );
      float z2 = rnd( prd_current.seed );
      float3 p;

      optix::cosine_sample_hemisphere( z1, z2, p );

      float scaling = roughness;

      p.x *= scaling;
      p.y *= scaling;

      p.z /= scaling;

      p = normalize( p );

      float3 v1, v2;
      float3 R = reflect( -w_v, surfel.normal );
      createONB( R, v1, v2 );

      prd_current.origin       = surfel.point;
      prd_current.direction    =  v1 * p.x + v2 * p.y + R * p.z;
      prd_current.attenuation *= F / reflectProb;
//      prd_current.countEmitted = true;

      prd_current.radiance = radiance;
      return;

    }


    //
    // absorb
    //
    prd_current.done = true;

  }

  prd_current.radiance = radiance;

} // closest_hit_bsdf



rtDeclareVariable( float3, emissionRadiance, , );

RT_PROGRAM
void
closest_hit_emission( )
{

  prd_current.radiance = prd_current.countEmitted ? emissionRadiance : make_float3( 0.f );
//  prd_current.radiance = emissionRadiance * prd_current.attenuation;
  prd_current.done = true;

}



/////////////////////////////////////////////////////////
/// \brief any_hit_occlusion
/////////////////////////////////////////////////////////
RT_PROGRAM
void
any_hit_occlusion( )
{

  // always opaque for now
  prd_shadow.attenuation = optix::make_float3( 0.0f );

  rtTerminateRay( );

}
