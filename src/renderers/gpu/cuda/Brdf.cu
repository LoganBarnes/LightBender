#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_stream_namespace.h>
#include "commonStructs.h"
#include "random.h"
#include "RendererObjects.hpp" // should be last to avoid FLT_MAX redefintion warning


// Create Orthonormal Basis from normalized vector
static
__device__ __inline__
void
createONB(
          const optix::float3 &n,
          optix::float3       &U,
          optix::float3       &V
          )
{
  using namespace optix;

  U = cross( n, make_float3( 0.0f, 1.0f, 0.0f ) );

  if ( dot( U, U ) < 1.e-3f )
  {
  U = cross( n, make_float3( 1.0f, 0.0f, 0.0f ) );
  }

  U = normalize( U );
  V = cross( n, U );
}



rtDeclareVariable( float3,               shading_normal,   attribute shading_normal, );
rtDeclareVariable( float3,               geometric_normal, attribute geometric_normal, );

rtDeclareVariable( PerRayData_pathtrace, prd_current,      rtPayload, );
rtDeclareVariable( PerRayData_shadow,    prd_shadow,       rtPayload, );

rtDeclareVariable( optix::Ray,           ray,              rtCurrentRay, );
rtDeclareVariable( float,                t_hit,            rtIntersectionDistance, );


rtDeclareVariable( unsigned int,         shadow_ray_type,  , );
rtDeclareVariable( float,                scene_epsilon,    , );
rtDeclareVariable( rtObject,             top_shadower,     , );

rtBuffer< Illuminator > illuminators;



/////////////////////////////////////////////////////////
/// \brief closest_hit_normals
///
///        Sets shading normal as the surface color
/////////////////////////////////////////////////////////
RT_PROGRAM
void
closest_hit_normals( )
{

  float3 worldGeoNormal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 worldShadeNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 ffnormal         = faceforward( worldShadeNormal, -ray.direction, worldGeoNormal );

  prd_current.radiance = ffnormal * 0.5f + 0.5f;
  prd_current.done     = true;

}



/////////////////////////////////////////////////////////
/// \brief closest_hit_simple_shading
/////////////////////////////////////////////////////////
RT_PROGRAM
void
closest_hit_simple_shading( )
{

  const float simpleShadeConstant = 0.8f;

  float3 worldGeoNormal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 worldShadeNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 ffnormal         = faceforward( worldShadeNormal, -ray.direction, worldGeoNormal );

  float3 radiance = make_float3( 0.0f );

  float3 hitPoint = ray.origin + t_hit * ray.direction;

  if ( prd_current.seed != static_cast< unsigned >( -1 ) )
  {

    prd_current.origin = hitPoint;

    float z1 = rnd( prd_current.seed );
    float z2 = rnd( prd_current.seed );
    float3 p;

    cosine_sample_hemisphere( z1, z2, p );

    float3 v1, v2;
    createONB( ffnormal, v1, v2 );

    prd_current.direction    = v1 * p.x + v2 * p.y + ffnormal * p.z;
    prd_current.attenuation *= make_float3( simpleShadeConstant );   // use the albedo as the diffuse response
    prd_current.countEmitted = false;

  }


  // loop vars
  float3 w_i;
  float distToLightPow2, distToLight;

  for ( int i = 0; i < illuminators.size( ); ++i )
  {

    Illuminator &illuminator = illuminators[ i ];

    float3 lightPos = illuminator.center;
    float3 flux     = illuminator.radiantFlux;


    // randomly sample sphere (only light shape for now)
    if ( prd_current.seed != static_cast< unsigned >( -1 ) )
    {

      float theta = rnd( prd_current.seed ) * 2 * M_PIf;
      float u     = rnd( prd_current.seed ) * 2.0 - 1.0;

      float xyCoeff = sqrt( 1.0 - u * u );

      float3 samplePos = make_float3(
                                     xyCoeff * cos( theta ),
                                     xyCoeff * sin( theta ),
                                     u
                                     );

      // sample on hemisphere in direction of point
      if ( dot( samplePos, normalize( hitPoint - lightPos ) ) < 0.0f )
      {

        samplePos = -samplePos;

      }

      lightPos += samplePos * illuminator.radius;

      // direction and distance to light
      w_i             = lightPos - hitPoint;
      distToLightPow2 = dot( w_i, w_i );
      distToLight     = sqrt( distToLightPow2 );
      w_i            /= distToLight; // normalizes w_i

      // lambertian emitter
      ///\todo: Sample by sollid angle for quicker convergance
      flux *= 0.5f * max( 0.0, dot( -w_i, samplePos ) );

    }
    else
    {

      // direction and distance to light
      w_i             = lightPos - hitPoint;
      distToLightPow2 = dot( w_i, w_i );
      distToLight     = sqrt( distToLightPow2 );
      w_i            /= distToLight; // normalizes w_i

      flux /= M_PIf;

    }


    float cosAngle = dot( ffnormal, w_i );

    if ( cosAngle > 0.0f )
    {

      // results from shadow ray
      PerRayData_shadow shadow_prd;
      shadow_prd.attenuation = optix::make_float3( 1.0f );

      // shadow ray
      optix::Ray shadow_ray(
                            hitPoint,
                            w_i,
                            shadow_ray_type,
                            scene_epsilon,
                            distToLight
                            );

      // shoot ray into scene
      rtTrace( top_shadower, shadow_ray, shadow_prd );

      float distPow2 = distToLight + illuminator.radius;
      distPow2 *= distPow2;

      radiance += ( simpleShadeConstant / M_PIf )          // lambertian pi normalization
                  * ( flux / ( M_PIf * 4.0f * distPow2 ) ) // inverse square law
                  * cosAngle                               // angle between normal and incident ray
                  * shadow_prd.attenuation;                // attenuation from shadowing objects

    }

  }

  prd_current.radiance = radiance;

} // closest_hit_simple_shading



rtDeclareVariable( float3, albedo,    , );
rtDeclareVariable( float,  roughness, , );

/////////////////////////////////////////////////////////
/// \brief closest_hit_bsdf
/////////////////////////////////////////////////////////
RT_PROGRAM
void
closest_hit_bsdf( )
{

//  float3 albedo   = make_float3( 0.13f ); // moon
// float3 albedo   = make_float3( 0.71f, 0.62f, 0.53f ); // clay
// float roughness = 0.3f;

  float3 worldGeoNormal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 worldShadeNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 ffnormal         = faceforward( worldShadeNormal, -ray.direction, worldGeoNormal );

  float3 radiance = make_float3( 0.0f );

  float3 hitPoint = ray.origin + t_hit * ray.direction;

  float3 w_o = -ray.direction;

  if ( prd_current.seed != static_cast< unsigned >( -1 ) )
  {

    prd_current.origin = hitPoint;

    float z1 = rnd( prd_current.seed );
    float z2 = rnd( prd_current.seed );
    float3 p;

    cosine_sample_hemisphere( z1, z2, p );

    float3 v1, v2;
    createONB( ffnormal, v1, v2 );

    prd_current.direction    = v1 * p.x + v2 * p.y + ffnormal * p.z;
    prd_current.attenuation *= albedo;  // use the albedo as the diffuse response
    prd_current.countEmitted = false;

    prd_current.attenuation = clamp( prd_current.attenuation, 0.0f, 1.0f );

  }

  // loop vars
  float3 w_i;
  float distToLightPow2, distToLight;

  for ( int i = 0; i < illuminators.size( ); ++i )
  {

    Illuminator &illuminator = illuminators[ i ];

    float3 lightPos = illuminator.center;
    float3 flux     = illuminator.radiantFlux;


    // randomly sample sphere (only light shape for now)
    if ( prd_current.seed != static_cast< unsigned >( -1 ) )
    {

      float theta = rnd( prd_current.seed ) * 2.0 * M_PIf;
      float u     = rnd( prd_current.seed ) * 2.0 - 1.0;

      float xyCoeff = sqrt( 1.0 - u * u );

      float3 samplePos = make_float3(
                                     xyCoeff * cos( theta ),
                                     xyCoeff * sin( theta ),
                                     u
                                     );

      // sample on hemisphere in direction of point
      if ( dot( samplePos, normalize( hitPoint - lightPos ) ) < 0.0f )
      {

        samplePos = -samplePos;

      }

      lightPos += samplePos * illuminator.radius;

      // direction and distance to light
      w_i             = lightPos - hitPoint;
      distToLightPow2 = dot( w_i, w_i );
      distToLight     = sqrt( distToLightPow2 );
      w_i            /= distToLight; // normalizes w_i

      // lambertian emitter
      ///\todo: Sample by sollid angle for quicker convergance
      flux *= 0.5f * max( 0.0, dot( -w_i, samplePos ) );

    }
    else
    {

      // direction and distance to light
      w_i             = lightPos - hitPoint;
      distToLightPow2 = dot( w_i, w_i );
      distToLight     = sqrt( distToLightPow2 );
      w_i            /= distToLight; // normalizes w_i

      ///\todo: Figure out correct term here
      flux /= M_PIf;

    }


    float cosAngle = dot( ffnormal, w_i );

    float3 localRadiance;

    if ( cosAngle > 0.0f )
    {

      // results from shadow ray
      PerRayData_shadow shadow_prd;
      shadow_prd.attenuation = optix::make_float3( 1.0f );

      // shadow ray
      optix::Ray shadow_ray(
                            hitPoint,
                            w_i,
                            shadow_ray_type,
                            scene_epsilon,
                            distToLight
                            );

      // shoot ray into scene
      rtTrace( top_shadower, shadow_ray, shadow_prd );

      float distPow2 = distToLight + illuminator.radius;
      distPow2 *= distPow2;

      // bsdf calculation added below
      localRadiance = ( flux / ( M_PIf * 4.0f * distPow2 ) ) // incident radiance
                      * cosAngle                             // angle between normal and incident ray
                      * shadow_prd.attenuation;              // attenuation from shadowing objects

      if ( dot( localRadiance, localRadiance ) > 1.0e-9f )
      {

        // brdf calculation

        // oren nayar diffuse brdf
        float gammaPow2 = roughness * roughness;

        float nDotL = optix::dot( ffnormal, w_i );
        float nDotV = optix::dot( ffnormal, w_o );

        float s = optix::dot( w_i, w_o ) - nDotL * nDotV;

        float t = s <= 0.0f ? 1.0f : max( nDotL, nDotV );

        float3 A = ( 1.0
                    - 0.5  * ( gammaPow2 / ( gammaPow2 + 0.33 ) )
                    + 0.17 * ( gammaPow2 / ( gammaPow2 + 0.13 ) ) * albedo
                    ) / M_PIf;

        float B = 0.45f * ( gammaPow2 / ( gammaPow2 + 0.09f ) ) / M_PIf;

        float3 onBrdf = albedo * ( A + B * s / t );

        radiance += localRadiance * onBrdf;

      }

    }

  }

  prd_current.radiance = radiance;

} // closest_hit_bsdf



rtDeclareVariable( float3, emissionRadiance, , );

RT_PROGRAM
void
closest_hit_emission( )
{

  prd_current.radiance = prd_current.countEmitted ? emissionRadiance : make_float3( 0.f );
  prd_current.done     = true;

}



/////////////////////////////////////////////////////////
/// \brief any_hit_occlusion
/////////////////////////////////////////////////////////
RT_PROGRAM
void
any_hit_occlusion( )
{

  // always opaque for now
  prd_shadow.attenuation = optix::make_float3( 0.0f );

  rtTerminateRay( );

}
