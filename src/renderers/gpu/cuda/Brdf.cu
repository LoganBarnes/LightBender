#include "hip/hip_runtime.h"
#include "optix.h"
#include "RendererObjects.hpp"


rtDeclareVariable( float3,              shading_normal,    attribute shading_normal, );
rtDeclareVariable( PerRayData_radiance, prd_radiance,      rtPayload, );


//
// Returns shading normal as the surface shading result
//
RT_PROGRAM
void
closest_hit_normals( )
{

  prd_radiance.result = normalize( rtTransformNormal(
                                                     RT_OBJECT_TO_WORLD,
                                                     shading_normal
                                                     ) ) * 0.5f + 0.5f;
}
