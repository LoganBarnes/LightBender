#include "hip/hip_runtime.h"
#include "optix/optix.h"
#include "RendererObjects.hpp"

rtDeclareVariable( float3,              shading_normal,    attribute shading_normal, );

rtDeclareVariable( PerRayData_radiance, prd_radiance,      rtPayload, );

rtDeclareVariable( optix::Ray,          ray,               rtCurrentRay, );
rtDeclareVariable( uint2,               launch_index,      rtLaunchIndex, );

rtDeclareVariable( unsigned int,        radiance_ray_type, , );
rtDeclareVariable( float,               scene_epsilon,     , );
rtDeclareVariable( rtObject,            top_object,        , );


//
// Pinhole camera implementation
//
rtDeclareVariable( float3, eye,       , );
rtDeclareVariable( float3, U,         , );
rtDeclareVariable( float3, V,         , );
rtDeclareVariable( float3, W,         , );

rtBuffer< float4, 2 >        output_buffer;

RT_PROGRAM
void
pinhole_camera( )
{

  size_t2 screen = output_buffer.size( );

  float2 d             = make_float2( launch_index ) / make_float2( screen ) * 2.f - 1.f;
  float3 ray_origin    = eye;
  float3 ray_direction = normalize( d.x * U + d.y * V + W );

  optix::Ray ray(
                 ray_origin,
                 ray_direction,
                 radiance_ray_type,
                 scene_epsilon
                 );

  PerRayData_radiance prd;
  prd.importance = 1.0f;
  prd.depth      = 0;

  rtTrace( top_object, ray, prd );

  output_buffer[ launch_index ] = make_float4( prd.result, 1.0 );

} // pinhole_camera



//
// Returns solid color for miss rays
//
rtDeclareVariable( float3, bg_color, , );

RT_PROGRAM
void
miss( )
{

  prd_radiance.result = bg_color;

}



//
// Returns shading normal as the surface shading result
//
RT_PROGRAM
void
closest_hit_radiance0( )
{

  prd_radiance.result = normalize( rtTransformNormal(
                                                     RT_OBJECT_TO_WORLD,
                                                     shading_normal
                                                     ) ) * 0.5f + 0.5f;
}



//
// Set pixel to solid color upon failure
//
rtDeclareVariable( float3, bad_color, , );

RT_PROGRAM
void
exception( )
{

  output_buffer[ launch_index ] = make_float4( bad_color, 1.0 );

}
